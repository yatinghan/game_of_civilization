#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <assert.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"
#include "circleBoxTest.cu_inl"
#include "exclusiveScan.cu_inl"
#include "cycleTimer.h"


#define BLOCK_WIDTH 32
#define BLOCK_HEIGHT 32
#define BLOCK_SIZE 1024

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct GlobalConstants {

    SceneName sceneName;

    int numCircles;
    float* position;
    float* velocity;
    float* color;
    float* radius;
    bool* isCircleInBlock;

    int imageWidth;
    int imageHeight;
    float* imageData;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int    cuConstNoiseYPermutationTable[256];
__constant__ int    cuConstNoiseXPermutationTable[256];
__constant__ float  cuConstNoise1DValueTable[256];

// color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float  cuConstColorRamp[COLOR_MAP_SIZE][3];


// including parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"

// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake() {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height-imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
//
// Update the position of the fireworks (if circle is firework)
__global__ void kernelAdvanceFireWorks() {
    const float dt = 1.f / 60.f;
    const float pi = 3.14159;
    const float maxDist = 0.25f;

    float* velocity = cuConstRendererParams.velocity;
    float* position = cuConstRendererParams.position;
    float* radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS) { // firework center; no update
        return;
    }

    // determine the fire-work center/spark indices
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i+1];

    // update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j+1] += velocity[index3j+1] * dt;

    // fire-work sparks
    float sx = position[index3j];
    float sy = position[index3j+1];

    // compute vector from firework-spark
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // compute distance from fire-work
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist) { // restore to starting position
        // random starting position on fire-work's rim
        float angle = (sfIdx * 2 * pi)/NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j+1] = position[index3i+1] + y;
        position[index3j+2] = 0.0f;

        // travel scaled unit length
        velocity[index3j] = cosA/5.0;
        velocity[index3j+1] = sinA/5.0;
        velocity[index3j+2] = 0.0f;
    }
}

// kernelAdvanceHypnosis
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis() {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles)
        return;

    float* radius = cuConstRendererParams.radius;

    float cutOff = 0.5f;
    // place circle back in center after reaching threshold radisus
    if (radius[index] > cutOff) {
        radius[index] = 0.02f;
    } else {
        radius[index] += 0.01f;
    }
}


// kernelAdvanceBouncingBalls
//
// Update the positino of the balls
__global__ void kernelAdvanceBouncingBalls() {
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    float* velocity = cuConstRendererParams.velocity;
    float* position = cuConstRendererParams.position;

    int index3 = 3 * index;
    // reverse velocity if center position < 0
    float oldVelocity = velocity[index3+1];
    float oldPosition = position[index3+1];

    if (oldVelocity == 0.f && oldPosition == 0.f) { // stop-condition
        return;
    }

    if (position[index3+1] < 0 && oldVelocity < 0.f) { // bounce ball
        velocity[index3+1] *= kDragCoeff;
    }

    // update velocity: v = u + at (only along y-axis)
    velocity[index3+1] += kGravity * dt;

    // update positions (only along y-axis)
    position[index3+1] += velocity[index3+1] * dt;

    if (fabsf(velocity[index3+1] - oldVelocity) < epsilon
        && oldPosition < 0.0f
        && fabsf(position[index3+1]-oldPosition) < epsilon) { // stop ball
        velocity[index3+1] = 0.f;
        position[index3+1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// move the snowflake animation forward one time step.  Updates circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float* positionPtr = &cuConstRendererParams.position[index3];
    float* velocityPtr = &cuConstRendererParams.velocity[index3];

    // loads from global memory
    float3 position = *((float3*)positionPtr);
    float3 velocity = *((float3*)velocityPtr);

    // hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // if the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ( (position.y + radius < 0.f) ||
         (position.x + radius) < -0.f ||
         (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // store updated positions and velocities to global memory
    *((float3*)positionPtr) = position;
    *((float3*)velocityPtr) = velocity;
}

// shadePixel -- (CUDA device code)
//
// given a pixel and a circle, determines the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
__device__ __inline__ void
shadePixel(int circleIndex, float2 pixelCenter, float3 p, float3 rgb, float rad, float4* imagePtr) {

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;
    float maxDist = rad * rad;

    // circle does not contribute to the image


    // float alpha = .5f;
    float alpha = pixelDist > maxDist ? 0.f : .5f;

    // there is a non-zero contribution.  Now compute the shading value

    // This conditional is in the inner loop, but it evaluates the
    // same direction for all threads so it's cost is not so
    // bad. Attempting to hoist this conditional is not a required
    // student optimization in Assignment 2
    if (cuConstRendererParams.sceneName == SNOWFLAKES || cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f-p.z);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

    } 

    float oneMinusAlpha = 1.f - alpha;

    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read

    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    // global memory write
    *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION
}



__device__ __inline__ void
shadeSnow(int circleIndex, float2 pixelCenter, float3 p, float3 rgb, float rad, float4* imagePtr) {

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;
    float maxDist = rad * rad;

    // circle does not contribute to the image
    if (pixelDist > maxDist) return;

    // there is a non-zero contribution.  Now compute the shading value

    // This conditional is in the inner loop, but it evaluates the
    // same direction for all threads so it's cost is not so
    // bad. Attempting to hoist this conditional is not a required
    // student optimization in Assignment 2

    const float kCircleMaxAlpha = .5f;
    const float falloffScale = 4.f;

    float normPixelDist = sqrt(pixelDist) / rad;
    rgb = lookupColor(normPixelDist);

    float maxAlpha = .6f + .4f * (1.f-p.z);
    maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
    float alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);


    float oneMinusAlpha = 1.f - alpha;

    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read

    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    // global memory write
    *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION
}

__global__ void kernelRenderPixels() {
    
    __shared__ float3 position[BLOCK_SIZE];
    __shared__ float3 color[BLOCK_SIZE];
    __shared__ float radius[BLOCK_SIZE];
    __shared__ uint inBlock[BLOCK_SIZE];
    __shared__ uint circlesCount[BLOCK_SIZE];
    __shared__ uint scratch[BLOCK_SIZE * 2];
    __shared__ uint intersectedCircles[BLOCK_SIZE];

    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    int numCircles = cuConstRendererParams.numCircles;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int indexInBlock = threadIdx.x*blockDim.y+threadIdx.y;
    int block_index = blockIdx.x * gridDim.y + blockIdx.y;
    int linearThreadIndex =  threadIdx.y * blockDim.x + threadIdx.x;
    

	for (int i = 0; i < numCircles; i+=BLOCK_SIZE) {
        int ic = i + indexInBlock;
        
        if (ic < numCircles && cuConstRendererParams.isCircleInBlock[block_index * numCircles + ic])  {
            inBlock[indexInBlock] = 1;     
            position[indexInBlock] = *(float3*)(&cuConstRendererParams.position[ic*3]);
            color[indexInBlock] = *(float3*)&(cuConstRendererParams.color[ic*3]);
            radius[indexInBlock] = cuConstRendererParams.radius[ic];
        } else {
            inBlock[indexInBlock] = 0;
        }
        
        __syncthreads();
        sharedMemExclusiveScan(linearThreadIndex, inBlock, circlesCount, scratch, BLOCK_SIZE);
        __syncthreads();
        if (ic < numCircles && inBlock[indexInBlock]) {
            intersectedCircles[circlesCount[indexInBlock]] = ic;
        }
        __syncthreads();

        if (col < imageWidth && row < imageHeight) {
            int totalCircles = circlesCount[BLOCK_SIZE - 1] + inBlock[BLOCK_SIZE - 1];
            for (int j = 0; j < totalCircles; j++) {
                int circle_index = intersectedCircles[j] - i;
                float invWidth = 1.f / imageWidth;
                float invHeight = 1.f / imageHeight;
                float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(col) + 0.5f),
                                                    invHeight * (static_cast<float>(row) + 0.5f));
                float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (row * imageWidth + col)]);
                shadePixel(circle_index, 
                        pixelCenterNorm, 
                        position[circle_index], 
                        color[circle_index], 
                        radius[circle_index], 
                        imgPtr);
            } 
        }
    }
}



__global__ void kernelRenderSnow() {
    
    __shared__ float3 position[BLOCK_SIZE];
    __shared__ float3 color[BLOCK_SIZE];
    __shared__ float radius[BLOCK_SIZE];
    __shared__ uint inBlock[BLOCK_SIZE];
    __shared__ uint circlesCount[BLOCK_SIZE];
    __shared__ uint scratch[BLOCK_SIZE * 2];
    __shared__ uint intersectedCircles[BLOCK_SIZE];

    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    int numCircles = cuConstRendererParams.numCircles;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int indexInBlock = threadIdx.x*blockDim.y+threadIdx.y;
    int block_index = blockIdx.x * gridDim.y + blockIdx.y;
    int linearThreadIndex =  threadIdx.y * blockDim.x + threadIdx.x;
    

	for (int i = 0; i < numCircles; i+=BLOCK_SIZE) {
        int ic = i + indexInBlock;
        
        if (ic < numCircles && cuConstRendererParams.isCircleInBlock[block_index * numCircles + ic])  {
            inBlock[indexInBlock] = 1;     
            position[indexInBlock] = *(float3*)(&cuConstRendererParams.position[ic*3]);
            color[indexInBlock] = *(float3*)&(cuConstRendererParams.color[ic*3]);
            radius[indexInBlock] = cuConstRendererParams.radius[ic];
        } else {
            inBlock[indexInBlock] = 0;
        }
        
        __syncthreads();
        sharedMemExclusiveScan(linearThreadIndex, inBlock, circlesCount, scratch, BLOCK_SIZE);
        __syncthreads();
        if (ic < numCircles && inBlock[indexInBlock]) {
            intersectedCircles[circlesCount[indexInBlock]] = ic;
        }
        __syncthreads();

        if (col < imageWidth && row < imageHeight) {
            int totalCircles = circlesCount[BLOCK_SIZE - 1] + inBlock[BLOCK_SIZE - 1];
            for (int j = 0; j < totalCircles; j++) {
                int circle_index = intersectedCircles[j] - i;
                float invWidth = 1.f / imageWidth;
                float invHeight = 1.f / imageHeight;
                float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(col) + 0.5f),
                                                    invHeight * (static_cast<float>(row) + 0.5f));
                float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (row * imageWidth + col)]);
                shadeSnow(circle_index, 
                    pixelCenterNorm, 
                    position[circle_index], 
                    color[circle_index], 
                    radius[circle_index], 
                    imgPtr);

            } 
        }
    }
}

/* 
 * for all circles, check if the circle intersects the block
 * each thread calculates a circle
 * @param ic: index of circle
 * @param ix: box row
 * @param iy: box column
 */
__global__ void kernelBlocksInCircles() {
    int ic;
    int row = (blockIdx.x * blockDim.x + threadIdx.x);
    int col = (blockIdx.y * blockDim.y + threadIdx.y);
    
    int imageHeight = cuConstRendererParams.imageHeight;
    int imageWidth = cuConstRendererParams.imageWidth;
    int numCircles = cuConstRendererParams.numCircles;

    if (blockIdx.x >= gridDim.x) return;
    if (blockIdx.y >= gridDim.y) return;
    
    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    float boxL = (static_cast<float>(col * BLOCK_WIDTH) + 0.5f) * invWidth ;
    float boxR = (static_cast<float>(min((col+1) * BLOCK_WIDTH, imageWidth)) + 0.5f) * invWidth;
    float boxB = (static_cast<float>(row * BLOCK_HEIGHT) +0.5f)* invHeight;
    float boxT = (static_cast<float>(min((row+1) * BLOCK_HEIGHT, imageHeight)) + 0.5f) * invHeight;    

    for (ic = 0; ic < numCircles; ic++) {
        float3 p = *(float3*)(&cuConstRendererParams.position[ic*3]);
        float rad = cuConstRendererParams.radius[ic];
        int isInBox = circleInBox(p.x, p.y, rad, boxL, boxR, boxT, boxB);
        cuConstRendererParams.isCircleInBlock[(row * (imageWidth/BLOCK_WIDTH) + col) * numCircles + ic] = isInBox; 
    }
}

__global__ void kernelCirclesInBlocks() {
    int index = blockIdx.x * blockDim.x + threadIdx.x; //circle index

    if (index >= cuConstRendererParams.numCircles)
        return;

    int index3 = 3 * index;

    // read position and radius
    float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
    float  rad = cuConstRendererParams.radius[index];

    // compute the bounding box of the circle. The bound is in integer
    // screen coordinates, so it's clamped to the edges of the screen.
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    int numCircles = cuConstRendererParams.numCircles;

    short minX = static_cast<short>(imageWidth * (p.x - rad));
    short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    short minY = static_cast<short>(imageHeight * (p.y - rad));
    short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

    // a bunch of clamps.  Is there a CUDA built-in for this?
    short screenMinX = max(min(minX, imageWidth), 0);
    short screenMaxX = max(min(maxX, imageWidth), 0);
    short screenMinY = max(min(minY, imageHeight), 0);
    short screenMaxY = max(min(maxY, imageHeight), 0);
    // short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
    // short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
    // short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
    // short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

    // iterate through the blocks that intersects the bounding box
    short leftmostBlock_col = screenMinX / BLOCK_WIDTH;
    short rightmostBlock_col= screenMaxX / BLOCK_WIDTH;
    short topBlock_row = screenMaxY / BLOCK_HEIGHT;
    short bottomBlock_row = screenMinY / BLOCK_HEIGHT;
    int blocksPerRow = (imageWidth + BLOCK_WIDTH - 1) / BLOCK_WIDTH;
    int blocksPerCol = (imageHeight + BLOCK_HEIGHT - 1) / BLOCK_HEIGHT;

    //block index : (r,c)
    for (short r = bottomBlock_row; r < blocksPerCol && r <= topBlock_row; r++) {
        for (short c = leftmostBlock_col; c < blocksPerRow && c <= rightmostBlock_col; c++) {
            // printf("block: %d, %d\n", r,c);
            
            cuConstRendererParams.isCircleInBlock[(r * blocksPerRow + c) * numCircles + index] = 1; 
        }
    }
}


////////////////////////////////////////////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {
    image = NULL;

    numCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
    cudaIsCircleInBlock = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

    if (position) {
        delete [] position;
        delete [] velocity;
        delete [] color;
        delete [] radius;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
        hipFree(cudaIsCircleInBlock);
    }
}

const Image*
CudaRenderer::getImage() {

    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    loadCircleScene(sceneName, numCircles, position, velocity, color, radius);
}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");

    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    int volume = sizeof(bool) * 
                 ((image->width+BLOCK_WIDTH-1)/BLOCK_WIDTH) *
                 ((image->height+BLOCK_HEIGHT-1)/BLOCK_HEIGHT) * numCircles;
    hipMalloc(&cudaIsCircleInBlock, volume);
    hipMemset(&cudaIsCircleInBlock, 0, volume);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numCircles = numCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;
    params.isCircleInBlock= cudaIsCircleInBlock;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int* permX;
    int* permY;
    float* value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // last, copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    } else {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void
CudaRenderer::advanceAnimation() {
     // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES) {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    } else if (sceneName == BOUNCING_BALLS) {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    } else if (sceneName == HYPNOSIS) {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    } else if (sceneName == FIREWORKS) {
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>();
    }
    hipDeviceSynchronize();
}

void
CudaRenderer::render() {
    // int numCircles = cuConstRendererParams.numCircles;
    dim3 blockDim(BLOCK_HEIGHT, BLOCK_WIDTH);
    dim3 gridDim((image->height+blockDim.x-1)/ blockDim.x,
                 (image->width+blockDim.y-1) / blockDim.y);
                
    dim3 blockDim1(256);
    dim3 gridDim1((numCircles+256-1)/256);
    kernelCirclesInBlocks<<<gridDim1, blockDim1>>>();
    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) 
        kernelRenderSnow<<<gridDim, blockDim>>>();
    else 
        kernelRenderPixels<<<gridDim, blockDim>>>();
}