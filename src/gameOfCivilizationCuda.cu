#include "hip/hip_runtime.h"
/**
 * Sequential Game of Civilization
 */

#include <iostream>
#include <cstdlib>
#include <unistd.h>
#include <fstream>
#include <string>
#include <sstream>
#include <vector>
#include <string>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <assert.h>

#include "gameOfCivilizationCuda.h"

#define BLOCK_WIDTH 32
#define BLOCK_HEIGHT 32
#define BLOCK_SIZE 1024
#define IDX(X, Y) ((X) * width + (Y))

using namespace std;


////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct GlobalConstants {

    int width;
    int height;
    int* grid;
    int* future;

};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;


////////////////////////////////////////////////////////////////////////////////////////


CudaGame::CudaGame(int W, int H, std::string filename) {
    this->width = W;
    int width = W;
    this->height = H;
    
    this->grid.resize(W*H);
    std::fill(this->grid.begin(), this->grid.end(), 0);
    this->future.resize(W*H);

    ifstream readfile(filename);
    if ( readfile.is_open() )
    {
        string fileline,xx,yy;
        int x, y;

        while (getline(readfile,fileline))
        {
            stringstream ss(fileline);
            getline(ss,xx,' ');
            getline(ss,yy,' ');
            x = stoi(xx);
            y = stoi(yy);
            this->grid[IDX(x, y)] = true;
        }
        setup();
    } 
    else {
        cout << "No such file, try again." << endl;
    }
}


void
CudaGame::setup() {

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");

    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDeviceGrid, sizeof(int) * width * height);
    hipMalloc(&cudaDeviceFuture, sizeof(int) * width * height);

    hipMemcpy(cudaDeviceGrid, &grid[0], sizeof(int) * width * height, hipMemcpyHostToDevice);
    //hipMemcpy(cudaDeviceFuture, &future[0], sizeof(int) * width * height, hipMemcpyHostToDevice);


    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.width = width;
    params.height = height;
    params.grid = cudaDeviceGrid;
    params.future = cudaDeviceFuture;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

}

void 
CudaGame::printGrid() {
    for (int i = 0; i < this->width; i++) 
    { 
        for (int j = 0; j < this->height; j++) 
        { 
            if (this->grid[IDX(i, j)] == 0) 
                cout << ". "; 
            else
                cout << "* "; 
        } 
        cout << endl; 
    } 
}

__global__ 
void kernelAdvanceGame() {
    
    //row?
    int l = blockIdx.x * blockDim.x + threadIdx.x;
    //col?
    int m = blockIdx.y * blockDim.y + threadIdx.y;
    int w = cuConstRendererParams.width;
    int width = cuConstRendererParams.width;
    int h = cuConstRendererParams.height;

    if (!(l < h && m < w && l > 0 && l < h-1 && m > 0 && m < w-1 )) 
        return;   //not in bound
    
    // finding no Of Neighbours that are alive 
    int aliveNeighbours = 0; 
    for (int i = -1; i <= 1; i++) 
        for (int j = -1; j <= 1; j++) 
            aliveNeighbours += cuConstRendererParams.grid[IDX(l + i, m + j)]; 

    // The cell needs to be subtracted from 
    // its neighbours as it was counted before 
    aliveNeighbours -= cuConstRendererParams.grid[IDX(l, m)]; 

    // Implementing the Rules of Life 

    // Cell is lonely and dies 
    if ((cuConstRendererParams.grid[IDX(l, m)] == 1) && (aliveNeighbours < 2)) 
        cuConstRendererParams.future[IDX(l, m)] = 0; 

    // Cell dies due to over population 
    else if ((cuConstRendererParams.grid[IDX(l, m)] == 1) && (aliveNeighbours > 3)) 
        cuConstRendererParams.future[IDX(l, m)] = 0; 

    // A new cell is born 
    else if ((cuConstRendererParams.grid[IDX(l, m)] == 0) && (aliveNeighbours == 3)) {
        cuConstRendererParams.future[IDX(l, m)] = 1; 
    }
        

    // Remains the same 
    else
        cuConstRendererParams.future[IDX(l, m)] = cuConstRendererParams.grid[IDX(l, m)]; 
    
    __syncthreads();

    // copy future to grid so that we don't have to re-copy
    cuConstRendererParams.grid[IDX(l, m)] = cuConstRendererParams.future[IDX(l, m)]; 
}

void
CudaGame::advanceGame() {
    
    dim3 blockDim(BLOCK_HEIGHT, BLOCK_WIDTH);
    dim3 gridDim((this->height+blockDim.x-1)/ blockDim.x,
                 (this->width+blockDim.y-1) / blockDim.y);
    // printf("grid dim: %d : %d\n", gridDim.x, gridDim.y);m
                
    kernelAdvanceGame<<<gridDim, blockDim>>>();
    
    hipMemcpy(&(this->future[0]),
               cudaDeviceFuture,
               sizeof(int) * width * height,
               hipMemcpyDeviceToHost);
        
    swap(grid, future);
}