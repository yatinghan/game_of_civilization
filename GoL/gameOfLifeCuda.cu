#include "hip/hip_runtime.h"
/**
 * Sequential Game of Civilization
 */

#include <iostream>
#include <cstdlib>
#include <unistd.h>
#include <fstream>
#include <string>
#include <sstream>
#include <vector>
#include <string>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <assert.h>

#include "gameOfLifeCuda.h"

#define BLOCK_WIDTH 32
#define BLOCK_HEIGHT 32
#define BLOCK_SIZE 1024
#define PIXEL_PER_THREAD 4
#define IDX(X, Y) ((X) * width + (Y))

using namespace std;


////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct GlobalConstants {

    int width;
    int height;
    int* grid;
    int* future;

};

__constant__ GlobalConstants cuConstRendererParams;


////////////////////////////////////////////////////////////////////////////////////////

CudaGame::CudaGame(int W, int H) {
    this->width = W;
    int width = W;
    this->height = H;
    
    this->grid.resize(W*H);
    std::fill(this->grid.begin(), this->grid.end(), 0);
    this->future.resize(W*H);
    for (int i = 0; i < H; i++) {
        for (int j = 0; j < W; j++) 
            if (i % 15 < 10 && j % 15 == 10) this->grid[IDX(i, j)] = true; //strips
    }
    setup();
}

CudaGame::CudaGame(int W, int H, std::string filename) {
    this->width = W;
    int width = W;
    this->height = H;
    
    this->grid.resize(W*H);
    std::fill(this->grid.begin(), this->grid.end(), 0);
    this->future.resize(W*H);

    ifstream readfile(filename);
    if ( readfile.is_open() )
    {
        string fileline,xx,yy;
        int x, y;

        while (getline(readfile,fileline))
        {
            stringstream ss(fileline);
            getline(ss,xx,' ');
            getline(ss,yy,' ');
            x = stoi(xx);
            y = stoi(yy);
            this->grid[IDX(x, y)] = true;
        }
        //setup();
    } 
    else {
        cout << "No such file, try again." << endl;
    }
}

void
CudaGame::setup() {

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);


    hipMalloc(&cudaDeviceGrid, sizeof(int) * width * height);
    hipMalloc(&cudaDeviceFuture, sizeof(int) * width * height);

    hipMemcpy(cudaDeviceGrid, &grid[0], sizeof(int) * width * height, hipMemcpyHostToDevice);

    GlobalConstants params;
    params.width = width;
    params.height = height;
    params.grid = cudaDeviceGrid;
    params.future = cudaDeviceFuture;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

}

void 
CudaGame::printGrid() {
    hipMemcpy(&(this->grid[0]),
               cudaDeviceFuture,
               sizeof(int) * width * height,
               hipMemcpyDeviceToHost);
    //swap(grid, future);
    for (int i = 0; i < this->width; i++) 
    { 
        for (int j = 0; j < this->height; j++) 
        { 
            if (this->grid[IDX(i, j)] == 0) 
                cout << ". "; 
            else
                cout << "* "; 
        } 
        cout << endl; 
    } 
}

__global__ 
void kernelAdvanceGame() {

    int start_idx = (blockIdx.x * blockDim.x + threadIdx.x) * PIXEL_PER_THREAD;
    int w = cuConstRendererParams.width;
    int h = cuConstRendererParams.height;
    int width = w;

    for (int idx = start_idx; idx < start_idx+PIXEL_PER_THREAD && idx < w*h; idx++) {
        int l = idx / h; //row ?
        int m = idx % h; //col ?
        if (l > 0 && l < h-1 && m > 0 && m < w-1) {

            // finding no Of Neighbours that are alive 
            int aliveNeighbours = 0; 
            for (int i = -1; i <= 1; i++) 
                for (int j = -1; j <= 1; j++) 
                    aliveNeighbours += (cuConstRendererParams.grid[IDX(l + i, m + j)] > 0); 

            // The cell needs to be subtracted from 
            // its neighbours as it was counted before 
            if (cuConstRendererParams.grid[IDX(l, m)] > 0) 
                cuConstRendererParams.grid[IDX(l, m)] = 1;

            __syncthreads();

            aliveNeighbours -= cuConstRendererParams.grid[IDX(l, m)]; 

            // Implementing the Rules of Life 

            // Cell is lonely and dies 
            if ((cuConstRendererParams.grid[IDX(l, m)] == 1) && (aliveNeighbours < 2)) 
                cuConstRendererParams.future[IDX(l, m)] = 0; 

            // Cell dies due to over population 
            else if ((cuConstRendererParams.grid[IDX(l, m)] == 1) && (aliveNeighbours > 3)) 
                cuConstRendererParams.future[IDX(l, m)] = 0; 

            // A new cell is born 
            else if ((cuConstRendererParams.grid[IDX(l, m)] == 0) && (aliveNeighbours == 3)) {
                cuConstRendererParams.future[IDX(l, m)] = 1; 
            }
                
            // Remains the same 
            else
                cuConstRendererParams.future[IDX(l, m)] = cuConstRendererParams.grid[IDX(l, m)]; 
        }
    }
    
}

__global__ 
void kernelSwap() {
    int start_idx = (blockIdx.x * blockDim.x + threadIdx.x) * (PIXEL_PER_THREAD );
    int dim = cuConstRendererParams.width * cuConstRendererParams.height;
    for (int idx = start_idx; idx < start_idx+PIXEL_PER_THREAD && idx < dim; idx++) {
            cuConstRendererParams.grid[idx] = cuConstRendererParams.future[idx]; 
    } 
}

void
CudaGame::advanceGame() {

    dim3 blockDim(BLOCK_SIZE);
    int dim = this->height*this->width;
    dim3 gridDim((dim / PIXEL_PER_THREAD + blockDim.x-1) / blockDim.x);
    kernelAdvanceGame<<<gridDim, blockDim>>>();
    kernelSwap<<<gridDim, blockDim>>>();
}

void
CudaGame::prepPrint() {
    hipMemcpy(&(this->grid[0]),
    this->cudaDeviceFuture,
    sizeof(int) * width * height,
    hipMemcpyDeviceToHost);
}